#include "hip/hip_runtime.h"
/*************************************************************************
 *  Copyright (c) 2016.
 *  All rights reserved.
 *  This file is part of the CLAIRE library.
 *
 *  CLAIRE is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  CLAIRE is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with CLAIRE.  If not, see <http://www.gnu.org/licenses/>.
 ************************************************************************/

#ifndef _DIFFERENTIATIONKERNEL_CPP_
#define _DIFFERENTIATIONKERNEL_CPP_

#include "DifferentiationKernel.hpp"
#include "cuda_helper.hpp"

#include "DifferentiationKernel.txx"

#include "nvToolsExt.h"

const uint32_t colors[] = { 0xff00ff00, 0xff0000ff, 0xffffff00, 0xffff00ff, 0xff00ffff, 0xffff0000, 0xffffffff };
const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define PUSH_RANGE(name,cid) { \
    int color_id = cid; \
    color_id = color_id%num_colors;\
    nvtxEventAttributes_t eventAttrib = {0}; \
    eventAttrib.version = NVTX_VERSION; \
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
    eventAttrib.colorType = NVTX_COLOR_ARGB; \
    eventAttrib.color = colors[color_id]; \
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
    eventAttrib.message.ascii = name; \
    nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();


using KernelUtils::SpectralKernelCallGPU;


namespace reg {

PetscErrorCode DifferentiationKernel::ScalarLaplacian(ScalarType b0) {
  //PUSH_RANGE("CU_Differentiation_ScalarLaplacian",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<NLaplacianKernel<1> >(nstart, nx, nl, 
    pXHat[0], b0*scale); CHKERRQ(ierr);
  //POP_RANGE
  PetscFunctionReturn(ierr);
}


PetscErrorCode DifferentiationKernel::LaplacianMod(ScalarType b0) {
  //PUSH_RANGE("CU_Differentiation_LaplacianMod",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<NLaplacianModKernel<1> >(nstart, nx, nl, 
    pXHat[0], pXHat[1], pXHat[2], 
    scale, b0); CHKERRQ(ierr);
  //POP_RANGE
  PetscFunctionReturn(ierr);
}
PetscErrorCode DifferentiationKernel::Laplacian(ScalarType b0, ScalarType b1) {
  //PUSH_RANGE("CU_Differentiation_Laplacian",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (b1 == 0.0) {
    ierr = SpectralKernelCallGPU<NLaplacianKernel<1> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale); CHKERRQ(ierr);
  } else {
    ierr = SpectralKernelCallGPU<RelaxedNLaplacianKernel<1> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, b1); CHKERRQ(ierr);
  }
  //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::LaplacianTol(ScalarType b0, ScalarType b1) {
  //PUSH_RANGE("CU_Differentiation_LaplacianTol",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ScalarType lognx = 0.;
  lognx += log2(static_cast<ScalarType>(nx[0]));
  lognx += log2(static_cast<ScalarType>(nx[1]));
  lognx += log2(static_cast<ScalarType>(nx[2]));
  
  KernelUtils::array3_t<ComplexType*> v;
  v.x = pXHat[0];
  v.y = pXHat[1];
  v.z = pXHat[2];
  
  if (b1 == 0.0) {
    ierr = SpectralKernelCallGPU<NLaplacianFilterKernel<1> >(nstart, nx, nl, v, 
      b0*scale, tol*lognx); CHKERRQ(ierr);
  } else {
    ierr = SpectralKernelCallGPU<RelaxedNLaplacianKernel<1> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, b1); CHKERRQ(ierr);
  }
  //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::Bilaplacian(ScalarType b0, ScalarType b1) {
  //PUSH_RANGE("CU_Differentiation_Bilaplacian",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (b1 == 0.0) {
    ierr = SpectralKernelCallGPU<NLaplacianKernel<2> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale); CHKERRQ(ierr);
  } else {
    ierr = SpectralKernelCallGPU<RelaxedNLaplacianKernel<2> >(nstart, nx, nl,
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, b1); CHKERRQ(ierr);
  }
  //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::Trilaplacian(ScalarType b0, ScalarType b1) {
  //PUSH_RANGE("CU_Differentiation_Trilaplacian",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (b1 == 0.0) {
    ierr = SpectralKernelCallGPU<NLaplacianKernel<3> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale); CHKERRQ(ierr);
  } else {
    ierr = SpectralKernelCallGPU<RelaxedNLaplacianKernel<3> >(nstart, nx, nl,
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, b1); CHKERRQ(ierr);
  }
  //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::TrilaplacianFunctional(ScalarType b0, ScalarType b1) {
  //PUSH_RANGE("CU_Differentiation_TrilaplacianFunctional",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = ThrowError("trilaplacian operator not implemented"); CHKERRQ(ierr);
  //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::InverseLaplacian(bool usesqrt, ScalarType b0, ScalarType b1) {
  //PUSH_RANGE("CU_Differentiation_InverseLaplacian",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (usesqrt) {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianSqrtKernel<1> >(nstart, nx, nl,
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianSqrtKernel<1> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  } else {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<1> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2], 
        scale, b0); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianKernel<1> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  }
  //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::InverseBilaplacian(bool usesqrt, ScalarType b0, ScalarType b1) {
  //PUSH_RANGE("CU_Differentiation_InverseBilaplacian",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (usesqrt) {
    if (b1 == 0.0) {
      /// scale/sqrt(b0*|lapik|^2) = scale/(sqrt(b0)*|lapik|)
      ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<1> >(nstart, nx, nl,
        pXHat[0], pXHat[1], pXHat[2],
        scale, sqrt(b0)); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianSqrtKernel<2> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  } else {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<2> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2], 
        scale, b0); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianKernel<2> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  }
  //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::InverseTrilaplacian(bool usesqrt, ScalarType b0, ScalarType b1) {
  //PUSH_RANGE("CU_Differentiation_InverseTrilaplacian",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (usesqrt) {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianSqrtKernel<3> >(nstart, nx, nl,
        pXHat[0], pXHat[1], pXHat[2],
        scale, sqrt(b0)); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianSqrtKernel<3> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  } else {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<3> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2], 
        scale, b0); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianKernel<3> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  }
  //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::Leray(ScalarType b0, ScalarType b1) {
  //PUSH_RANGE("CU_Differentiation_Leray",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<LerayKernel>(nstart, nx, nl, 
    pXHat[0], pXHat[1], pXHat[2], 
    scale, b0, b1); CHKERRQ(ierr);
  //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::InvRegLeray(ScalarType b0, ScalarType b1, ScalarType b2) {
  //PUSH_RANGE("CU_Differentiation_InvRegLeray",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<LerayKernel>(nstart, nx, nl, 
    pXHat[0], pXHat[1], pXHat[2], 
    scale, b0, b1); CHKERRQ(ierr);
  ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<1> >(nstart, nx, nl, 
    pXHat[0], pXHat[1], pXHat[2], 
    1., b2); CHKERRQ(ierr);
  //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::GaussianFilter(const ScalarType c[3]) {
  //PUSH_RANGE("CU_Differentiation_GaussianFilter",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<GaussianFilterKernel>(nstart, nx, nl, 
    pXHat[0], c[0], c[1], c[2], scale); CHKERRQ(ierr);
  //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::Gradient() {
  //PUSH_RANGE("CU_Differentiation_Gradient",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<GradientKernel>(nstart, nx, nl, 
    pXHat[0], pXHat[1], pXHat[2], scale); CHKERRQ(ierr);
  //POP_RANGE
  PetscFunctionReturn(ierr);
}
PetscErrorCode DifferentiationKernel::Divergence() {
  //PUSH_RANGE("CU_Differentiation_Divergence",2)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<DivergenceKernel>(nstart, nx, nl, 
    pXHat[0], pXHat[1], pXHat[2], scale); CHKERRQ(ierr);
  //POP_RANGE
  PetscFunctionReturn(ierr);
}

} // namespace reg

#endif
