#include "hip/hip_runtime.h"
/*************************************************************************
 *  Copyright (c) 2016.
 *  All rights reserved.
 *  This file is part of the CLAIRE library.
 *
 *  CLAIRE is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  CLAIRE is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with CLAIRE.  If not, see <http://www.gnu.org/licenses/>.
 ************************************************************************/

#ifndef _CLAIREUTILSKERNEL_CU_
#define _CLAIREUTILSKERNEL_CU_

#include "CLAIREUtils.hpp"
#include "cuda_helper.hpp"

#include "nvToolsExt.h"

const uint32_t colors[] = { 0xff00ff00, 0xff0000ff, 0xffffff00, 0xffff00ff, 0xff00ffff, 0xffff0000, 0xffffffff };
const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define PUSH_RANGE(name,cid) { \
    int color_id = cid; \
    color_id = color_id%num_colors;\
    nvtxEventAttributes_t eventAttrib = {0}; \
    eventAttrib.version = NVTX_VERSION; \
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
    eventAttrib.colorType = NVTX_COLOR_ARGB; \
    eventAttrib.color = colors[color_id]; \
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
    eventAttrib.message.ascii = name; \
    nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();

// CUDA kernel to evaluate point-wise norm of a vector field
__global__ void VecFieldPointWiseNormKernel(ScalarType *p_m, const ScalarType *p_X1, const ScalarType *p_X2, const ScalarType *p_X3, IntType nl) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i < nl) {
        p_m[i] = sqrtf(p_X1[i]*p_X1[i] + p_X2[i]*p_X2[i] + p_X3[i]*p_X3[i]);
    }
}

__global__ void CopyStridedToFlatVecKernel(ScalarType *pX, const ScalarType *p_x1, const ScalarType *p_x2, const ScalarType *p_x3, IntType nl) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < nl) {
        pX[3*i + 0] = p_x1[i];
        pX[3*i + 1] = p_x2[i];
        pX[3*i + 2] = p_x3[i];
    }

}

__global__ void CopyStridedFromFlatVecKernel(ScalarType *p_x1, ScalarType *p_x2, ScalarType *p_x3, const ScalarType* pX, IntType nl) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < nl) {
        p_x1[i] = pX[3*i + 0];
        p_x2[i] = pX[3*i + 1];
        p_x3[i] = pX[3*i + 2];
    }
}

__global__ void SetValueKernel(ScalarType* p, ScalarType v, IntType nl) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < nl) {
        p[i] = v;
    }
}


namespace reg {
  
  
PetscErrorCode SetValue(ScalarType* p, ScalarType v, IntType nl) {
  //PUSH_RANGE("CU_UtilsKernel_SetValue",3)
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  dim3 block(256, 1, 1);
  dim3 grid((nl + 255)/256, 1, 1);
  
  SetValueKernel<<<grid, block>>>(p, v, nl);
  hipDeviceSynchronize();
  cudaCheckKernelError();
  //POP_RANGE
  PetscFunctionReturn(ierr);
}
  

/********************************************************************
 * @brief compute pointwise norm of vector field
 *******************************************************************/
PetscErrorCode VecFieldPointWiseNormGPU(ScalarType* p_m, const ScalarType* p_X1, const ScalarType* p_X2, const ScalarType* p_X3, IntType nl) {
    //PUSH_RANGE("CU_UtilsKernel_VecFieldPointWiseNormGPU",3)
    PetscErrorCode ierr = 0;
    PetscFunctionBegin;
    
    dim3 block(256, 1, 1);
    dim3 grid((nl + 255)/256, 1, 1);
    
    VecFieldPointWiseNormKernel<<<grid, block>>>(p_m, p_X1, p_X2, p_X3, nl);
    hipDeviceSynchronize();
    cudaCheckKernelError();
    //POP_RANGE
    PetscFunctionReturn(ierr);

}


/********************************************************************
 * @brief Copy vector field to a flat array in strided fashion
 *******************************************************************/
PetscErrorCode CopyStridedToFlatVec(ScalarType* pX, const ScalarType* p_x1, const ScalarType* p_x2, const ScalarType* p_x3, IntType nl) {
    //PUSH_RANGE("CU_UtilsKernel_CopyStridedToFlatVec",3)
    PetscFunctionBegin;
    PetscErrorCode ierr = 0;
    
    int threads = 256;
    int blocks = (nl + 255)/threads;

    CopyStridedToFlatVecKernel<<<blocks,threads>>>(pX, p_x1, p_x2, p_x3, nl);
    hipDeviceSynchronize();
    cudaCheckKernelError();
    //POP_RANGE
    PetscFunctionReturn(ierr);
}

/********************************************************************
 * @brief Copy vector field to a flat array in strided fashion
 *******************************************************************/
PetscErrorCode CopyStridedFromFlatVec(ScalarType* p_x1, ScalarType* p_x2, ScalarType* p_x3, const ScalarType* pX, IntType nl) {
    //PUSH_RANGE("CU_UtilsKernel_CopyStridedFromFlatVec",3)
    PetscFunctionBegin;
    PetscErrorCode ierr = 0;
    
    int threads = 256;
    int blocks = (nl + 255)/threads;

    CopyStridedFromFlatVecKernel<<<blocks,threads>>>(p_x1, p_x2, p_x3, pX, nl);
    hipDeviceSynchronize();
    cudaCheckKernelError();
    //POP_RANGE
    PetscFunctionReturn(ierr);
}

}

#endif
