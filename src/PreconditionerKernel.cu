#include "hip/hip_runtime.h"
/*************************************************************************
 *  Copyright (c) 2018.
 *  All rights reserved.
 *  This file is part of the CLAIRE library.
 *
 *  CLAIRE is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  CLAIRE is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with CLAIRE. If not, see <http://www.gnu.org/licenses/>.
 ************************************************************************/

#include "PreconditionerKernel.hpp"
#include "cuda_helper.hpp"

#include "PreconditionerKernel.txx"

#include "nvToolsExt.h"

const uint32_t colors[] = { 0xff00ff00, 0xff0000ff, 0xffffff00, 0xffff00ff, 0xff00ffff, 0xffff0000, 0xffffffff };
const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define PUSH_RANGE(name,cid) { \
    int color_id = cid; \
    color_id = color_id%num_colors;\
    nvtxEventAttributes_t eventAttrib = {0}; \
    eventAttrib.version = NVTX_VERSION; \
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
    eventAttrib.colorType = NVTX_COLOR_ARGB; \
    eventAttrib.color = colors[color_id]; \
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
    eventAttrib.message.ascii = name; \
    nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();

namespace reg {
  
using KernelUtils::KernelCallGPU;
using KernelUtils::ReductionKernelCallGPU;

PetscErrorCode H0PrecondKernel::gMgMT2 () {
 
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
   //PUSH_RANGE("CU_PrecondKernel_gMgMT2",4)
  
  ierr = KernelCallGPU<H0Kernel2>(nl, 
                                 pM[0], pM[1], pM[2], 
                                 pVhat[0], pVhat[1], pVhat[2], 
                                 pGmt[0], pGmt[1], pGmt[2]); CHKERRQ(ierr);
  //POP_RANGE  
  PetscFunctionReturn(ierr);
}
PetscErrorCode H0PrecondKernel::res2 (ScalarType &res) {

  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
    //PUSH_RANGE("CU_PrecondKernel_res2",4)
  
  ierr = ReductionKernelCallGPU<H0Kernel2>(res, pWS, nl, 
                                          pM[0], pM[1], pM[2],
                                          pP[0], pP[1], pP[2],
                                          pRes[0], pRes[1], pRes[2],
                                          pGmt[0], pGmt[1], pGmt[2],
                                          diag); CHKERRQ(ierr);
  //POP_RANGE  
  PetscFunctionReturn(ierr);
}
PetscErrorCode H0PrecondKernel::pTAp2 (ScalarType &res) {

  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
    //PUSH_RANGE("CU_PrecondKernel_pTAp2",4)
  
  ierr = ReductionKernelCallGPU<H0Kernel2>(res, pWS, nl, 
                                          pM[0], pM[1], pM[2],
                                          pP[0], pP[1], pP[2],
                                          pGmt[0], pGmt[1], pGmt[2],
                                          diag); CHKERRQ(ierr);
  //POP_RANGE  
  PetscFunctionReturn(ierr);
}

PetscErrorCode H0PrecondKernel::gMgMT () {

  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
    //PUSH_RANGE("CU_PrecondKernel_gMgMT",4)
  
  ierr = KernelCallGPU<H0Kernel>(nl, 
                                 pM[0], pM[1], pM[2], 
                                 pVhat[0], pVhat[1], pVhat[2], 
                                 pGmt[0], pGmt[1], pGmt[2]); CHKERRQ(ierr);
  //POP_RANGE  
  PetscFunctionReturn(ierr);
}

PetscErrorCode H0PrecondKernel::res (ScalarType &res) {

  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
    //PUSH_RANGE("CU_PrecondKernel_res",4)
  
  ierr = ReductionKernelCallGPU<H0Kernel>(res, pWS, nl, 
                                          pM[0], pM[1], pM[2],
                                          pP[0], pP[1], pP[2],
                                          pRes[0], pRes[1], pRes[2],
                                          pVhat[0], pVhat[1], pVhat[2]); CHKERRQ(ierr);
  // POP_RANGE 
  PetscFunctionReturn(ierr);
}

PetscErrorCode H0PrecondKernel::pTAp (ScalarType &res) {

  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
    //PUSH_RANGE("CU_PrecondKernel_pTAp",4)
  
  ierr = ReductionKernelCallGPU<H0Kernel>(res, pWS, nl, 
                                          pM[0], pM[1], pM[2],
                                          pP[0], pP[1], pP[2]); CHKERRQ(ierr);
   // POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode H0PrecondKernel::CGres (ScalarType &res) {

  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
   // PUSH_RANGE("CU_PrecondKernel_CGres",4)
  
  ScalarType alpha = res;
  
  ierr = ReductionKernelCallGPU<H0KernelCG>(res, pWS, nl, 
                                            pM[0], pM[1], pM[2],
                                            pP[0], pP[1], pP[2],
                                            pRes[0], pRes[1], pRes[2],
                                            pVhat[0], pVhat[1], pVhat[2],
                                            alpha); CHKERRQ(ierr);
   // POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode H0PrecondKernel::CGp (ScalarType alpha) {
 
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  // PUSH_RANGE("CU_PrecondKernel_CGp",4)
    
  ierr = KernelCallGPU<H0KernelCG>(nl, 
                                   pP[0], pP[1], pP[2],
                                   pRes[0], pRes[1], pRes[2],
                                   alpha); CHKERRQ(ierr);
    //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode CFLStatKernel::CFLx (ScalarType &ratio) {
 
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
   //PUSH_RANGE("CU_CFLStatKernel_CFLx",4)
  
  ScalarType res;
    
  ierr = ReductionKernelCallGPU<CFLKernel>(res, nl, pV[0], h, dt); CHKERRQ(ierr);
  
  //ratio = res/ng;
  
  ratio = res;
    //POP_RANGE
  PetscFunctionReturn(ierr);
}

PetscErrorCode H0PrecondKernel::Norm (ScalarType &norm) {
  
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
   //PUSH_RANGE("CU_PrecondKernel_Norm",4)
  
  ScalarType res;
    
  ierr = ReductionKernelCallGPU<NormKernel>(res, nl, pGmt[0]); CHKERRQ(ierr);
  
  //ratio = res/ng;
  
  norm = res;
   // POP_RANGE
  PetscFunctionReturn(ierr);
}

} // namespace reg
